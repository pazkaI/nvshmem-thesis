#include "hip/hip_runtime.h"
#include <iostream>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

const int IMAGE_WIDTH = 960;
const int IMAGE_HEIGHT = 960;
const int IMAGE_SIZE = IMAGE_WIDTH * IMAGE_HEIGHT;

const int RAY_ARRAY_LENGTH = 2048;
const int ITERATIONS = 110;

__device__
int indexAt(int x, int y) {
    return x * IMAGE_HEIGHT + y;
}

__global__
void setupRandom(hiprandState *random, int seed) {
    hiprand_init(seed, 0, 0, random);
}

__global__
void initRayArray(float *rayArray, hiprandState* random, int worldRank) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < IMAGE_SIZE) {
        for (int i = 0; i < RAY_ARRAY_LENGTH; i++) {
            rayArray[index * RAY_ARRAY_LENGTH + i] = hiprand_uniform(random);
        }
    }
}

__global__
void compute(float *image, const float *rayArray, int worldRank, int sliceWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x / sliceWidth == worldRank) {
        float accumulated = 0;
        for (int i = 0; i < RAY_ARRAY_LENGTH; i++) {
            accumulated += rayArray[indexAt(x,y) * RAY_ARRAY_LENGTH + i];
        }

        image[indexAt(x,y)] = accumulated;
    } else {
        image[indexAt(x,y)] = 0;
    }
}

__global__
void computeSend(float *image, float *image2, float *slices, const float *rayArray, int worldRank, int sliceWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // send image
    int destRank = x / sliceWidth;
    nvshmemx_float_put_nbi_block(&slices[indexAt(worldRank * sliceWidth + x % sliceWidth, 0)], &image[indexAt(x, 0)], IMAGE_HEIGHT, destRank);

    // compute new image
    if (x / sliceWidth == worldRank) {
        float accumulated = 0;
        for (int i = 0; i < RAY_ARRAY_LENGTH; i++) {
            accumulated += rayArray[indexAt(x,y) * RAY_ARRAY_LENGTH + i];
        }

        image2[indexAt(x,y)] = accumulated;
    } else {
        image2[indexAt(x,y)] = 0;
    }
}

__global__
void send(float *image, float *slices, int worldRank, int sliceWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int destRank = x / sliceWidth;

    nvshmemx_float_put_nbi_block(&slices[indexAt(worldRank * sliceWidth + x % sliceWidth, 0)], &image[indexAt(x, 0)], IMAGE_HEIGHT, destRank);
}

__global__
void compositeSend(const float *slices, float *compositedSlice, float *result, int worldSize, int worldRank, int sliceWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    for (int i = 0; i < worldSize; i++) {
        sum += slices[indexAt(i * sliceWidth + x, y)];
    }

    compositedSlice[indexAt(x,y)] = sum;
    nvshmemx_float_put_block(&result[indexAt(worldRank * sliceWidth + x, 0)], &compositedSlice[indexAt(x, 0)], IMAGE_HEIGHT, 0);
}

__global__
void printTimings(const float *maxExecutionTime) {
    printf("execution done on rank 0.\n max execution time: %f ms\n", *maxExecutionTime);
}

int main(int argc, char *argv[]) {

    nvshmem_init();

    int worldSize = nvshmem_n_pes();
    int worldRank = nvshmem_my_pe();

    int nodeRank = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    CUDA_SAFE_CALL(hipSetDevice(nodeRank));

    hipStream_t stream;
    CUDA_SAFE_CALL(hipStreamCreate(&stream));

    int sliceSize = IMAGE_SIZE / worldSize;

    auto *result = (float *) nvshmem_malloc(IMAGE_SIZE * sizeof(float));

    auto *image = (float *) nvshmem_malloc(IMAGE_SIZE * sizeof(float));
    CUDA_SAFE_CALL(hipMemset(image, 0, IMAGE_SIZE * sizeof(float)));

    auto *image2 = (float *) nvshmem_malloc(IMAGE_SIZE * sizeof(float));
    CUDA_SAFE_CALL(hipMemset(image2, 0, IMAGE_SIZE * sizeof(float)));

    auto *slices = (float *) nvshmem_malloc(IMAGE_SIZE * sizeof(float));
    auto *compositedSlice = (float *) nvshmem_malloc(sliceSize * sizeof(float));

    float *rayArray;
    CUDA_SAFE_CALL(hipMalloc(&rayArray, IMAGE_SIZE * RAY_ARRAY_LENGTH * sizeof(float)));

    // create pseudo-random number generator
    hiprandState *random;
    CUDA_SAFE_CALL(hipMalloc(&random, sizeof(hiprandState)));
    setupRandom<<<1, 1, 0, stream>>>(random, worldRank);

    initRayArray<<<std::ceil((float) IMAGE_SIZE / 1024), 1024, 0, stream>>>(rayArray, random, worldRank);
    CUDA_SAFE_CALL(hipStreamSynchronize(stream));

    hipEvent_t start, end;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&end));

    nvshmem_barrier_all();

    int sliceWidth = IMAGE_WIDTH / worldSize;

    dim3 threads(1, IMAGE_HEIGHT, 1);
    dim3 imageGrid(IMAGE_WIDTH, 1, 1);
    dim3 sliceGrid(sliceWidth, 1, 1);

    compute<<<imageGrid, threads, 0, stream>>>(image, rayArray, worldRank, sliceWidth);

    for (int i = 0; i < ITERATIONS; i++) {

        if (i == 10) {
            CUDA_SAFE_CALL(hipEventRecord(start, stream));
        }

        computeSend<<<imageGrid, threads, 0, stream>>>(image, image2, slices, rayArray, worldRank, sliceWidth);
        nvshmemx_quiet_on_stream(stream);
        nvshmemx_barrier_all_on_stream(stream);

        compositeSend<<<sliceGrid, threads, 0, stream>>>(slices, compositedSlice, result, worldSize, worldRank, sliceWidth);

        std::swap(image, image2);
    }

    send<<<imageGrid, threads, 0, stream>>>(image, slices, worldRank, sliceWidth);
    nvshmemx_quiet_on_stream(stream);
    nvshmemx_barrier_all_on_stream(stream);

    compositeSend<<<sliceGrid, threads, 0, stream>>>(slices, compositedSlice, result, worldSize, worldRank, sliceWidth);

    // done.

    CUDA_SAFE_CALL(hipEventRecord(end, stream));
    CUDA_SAFE_CALL(hipEventSynchronize(end));

    nvshmem_barrier_all();

    float executionTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&executionTime, start, end));

    auto *maxExecutionTime = (float *) nvshmem_malloc(sizeof(float));
    CUDA_SAFE_CALL(hipMemcpy(maxExecutionTime, &executionTime, sizeof(float), hipMemcpyHostToDevice));
    nvshmemx_float_max_reduce_on_stream(NVSHMEM_TEAM_WORLD, maxExecutionTime, maxExecutionTime, 1, stream);

    if (worldRank == 0) {
        printTimings<<<1, 1, 0, stream>>>(maxExecutionTime);
    }
    hipStreamSynchronize(stream);

    nvshmem_free(result);
    nvshmem_free(image);
    nvshmem_free(slices);
    nvshmem_free(compositedSlice);

    CUDA_SAFE_CALL(hipFree(rayArray));

    nvshmem_finalize();
    return 0;
}