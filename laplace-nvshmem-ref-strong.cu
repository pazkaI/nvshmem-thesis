#include "hip/hip_runtime.h"
#include <iostream>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <chrono>
#include <hiprand/hiprand_kernel.h>

#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)

const int WIDTH = 512;
const int HEIGHT = 512;
const int DEPTH = 512;
const int SIZE = WIDTH * HEIGHT * DEPTH;

const int ITERATIONS = 110;

__device__
int indexAt(int x, int y, int z) {
    return WIDTH * HEIGHT * z + WIDTH * y + x;
}

__device__
int indexAtWithPadding(int x, int y, int z) {
    return (WIDTH + 2) * (HEIGHT + 2) * z + (WIDTH + 2) * y + x;
}

__device__
bool doubleEquals(double a, double b) {
    return fabs(a - b) < 1e-8;
}

__global__
void setupRandom(hiprandState *random, int seed) {
    hiprand_init(seed, 0, 0, random);
}

__global__
void initSlice(double *slice, hiprandState *random, dim3 max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (z < max.z && y < max.y && x < max.x) {

        // adjust for halo padding
        z += 1;
        y += 1;
        x += 1;

        slice[indexAtWithPadding(x, y, z)] = hiprand_uniform(random);
    }
}

__global__
void removePadding(const double *slice, double *unpaddedSlice, dim3 max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (z < max.z && y < max.y && x < max.x) {
        unpaddedSlice[indexAt(x, y, z)] = slice[indexAtWithPadding(x + 1, y + 1, z + 1)];
    }
}

__global__
void laplaceInnerKernel(const double *slice, double *resultSlice, dim3 max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // adjust for halo padding and inner computation
    z += 2;
    y += 1;
    x += 1;

    double sum = -6 * slice[indexAtWithPadding(x, y, z)];
    sum += slice[indexAtWithPadding(x - 1, y, z)];
    sum += slice[indexAtWithPadding(x + 1, y, z)];
    sum += slice[indexAtWithPadding(x, y - 1, z)];
    sum += slice[indexAtWithPadding(x, y + 1, z)];
    sum += slice[indexAtWithPadding(x, y, z - 1)];
    sum += slice[indexAtWithPadding(x, y, z + 1)];

    resultSlice[indexAtWithPadding(x, y, z)] += 0.01 * sum;
}

__global__
void laplaceOuterKernel(const double *slice, double *resultSlice, dim3 max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // move to z=0 and z=max.z-1
    if (z == 1) {
        z = max.z - 1;
    }

    // adjust for halo padding
    z += 1;
    y += 1;
    x += 1;

    double sum = -6 * slice[indexAtWithPadding(x, y, z)];
    sum += slice[indexAtWithPadding(x - 1, y, z)];
    sum += slice[indexAtWithPadding(x + 1, y, z)];
    sum += slice[indexAtWithPadding(x, y - 1, z)];
    sum += slice[indexAtWithPadding(x, y + 1, z)];
    sum += slice[indexAtWithPadding(x, y, z - 1)];
    sum += slice[indexAtWithPadding(x, y, z + 1)];

    resultSlice[indexAtWithPadding(x, y, z)] += 0.01 * sum;
}

__global__
void sync_neighbours(int worldRank, int worldSize, uint64_t *commSync, int counter) {

    // notify neighboring gpus
    if (worldRank != 0) {
        nvshmemx_signal_op(commSync, counter, NVSHMEM_SIGNAL_SET, worldRank - 1);
    }

    if (worldRank != worldSize - 1) {
        nvshmemx_signal_op(commSync + 1, counter, NVSHMEM_SIGNAL_SET, worldRank + 1);
    }



    // wait for own signal
    if (worldRank != 0) {
        nvshmem_uint64_wait_until(commSync + 1, NVSHMEM_CMP_GE, counter);
    }

    if (worldRank != worldSize - 1) {
        nvshmem_uint64_wait_until(commSync, NVSHMEM_CMP_GE, counter);
    }
}


__global__
void laplace_singleGPU(double *input, const double *buffer, dim3 max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (z < max.z && y < max.y && x < max.x) {

        double sum = -6 * buffer[indexAt(x, y, z)];

        if (x - 1 >= 0) {
            sum += buffer[indexAt(x - 1, y, z)];
        }

        if (x + 1 < WIDTH) {
            sum += buffer[indexAt(x + 1, y, z)];
        }

        if (y - 1 >= 0) {
            sum += buffer[indexAt(x, y - 1, z)];
        }

        if (y + 1 < HEIGHT) {
            sum += buffer[indexAt(x, y + 1, z)];
        }

        if (z - 1 >= 0) {
            sum += buffer[indexAt(x, y, z - 1)];
        }

        if (z + 1 < DEPTH) {
            sum += buffer[indexAt(x, y, z + 1)];
        }


        input[indexAt(x, y, z)] += 0.01 * sum;
    }
}

__global__
void verify(const double *a, const double *b, dim3 max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (z < max.z && y < max.y && x < max.x) {
        if (!doubleEquals(a[indexAt(x, y, z)], b[indexAt(x, y, z)])) {
            printf("computation mismatch at index x=%d,y=%d,z=%d: %f vs %f!\n", x, y, z, a[indexAt(x, y, z)], b[indexAt(x, y, z)]);
        }
    }
}

__global__
void printTimings(float *maxExecutionTime) {
    printf("computation done on rank 0.\n max execution time: %f ms\n", *maxExecutionTime);
}

int main(int argc, char *argv[]) {

    bool verifyResults = false;
    if (argc > 1) {
        if (strcmp(argv[1], "1") == 0) {
            verifyResults = true;
        }
    }

    nvshmem_init();

    int worldSize = nvshmem_n_pes();
    int worldRank = nvshmem_my_pe();

    int nodeRank = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    CUDA_SAFE_CALL(hipSetDevice(nodeRank));

    hipStream_t stream;
    CUDA_SAFE_CALL(hipStreamCreate(&stream));

    dim3 sliceSize, paddedSliceSize, grid, threads;
    sliceSize.x = WIDTH;
    sliceSize.y = HEIGHT;
    sliceSize.z = DEPTH / worldSize;

    paddedSliceSize.x = sliceSize.x + 2;
    paddedSliceSize.y = sliceSize.y + 2;
    paddedSliceSize.z = sliceSize.z + 2;

    threads.x = WIDTH;
    threads.y = 1;
    threads.z = 1;
    grid.x = std::ceil((float) sliceSize.x / threads.x);
    grid.y = std::ceil((float) sliceSize.y / threads.y);
    grid.z = std::ceil((float) sliceSize.z / threads.z);

    // inner and outer kernel grids
    dim3 innerGrid, outerGrid;
    innerGrid.x = std::ceil((float) sliceSize.x / threads.x);
    innerGrid.y = std::ceil((float) sliceSize.y / threads.y);
    innerGrid.z = std::ceil((float) (sliceSize.z - 2) / threads.z);

    outerGrid.x = std::ceil((float) sliceSize.x / threads.x);
    outerGrid.y = std::ceil((float) sliceSize.y / threads.y);
    outerGrid.z = std::ceil((float) 2 / threads.z);

    if (worldRank == 0) {
        printf("computation start: \n gridDim: %d, %d, %d\n threadDim: %d, %d, %d\n", grid.x, grid.y, grid.z, threads.x, threads.y, threads.z);
    }

    size_t sliceSizeMemWithPadding = paddedSliceSize.x * paddedSliceSize.y * paddedSliceSize.z * sizeof(double);

    // device memory
    auto *slice = (double *) nvshmem_malloc(sliceSizeMemWithPadding);
    CUDA_SAFE_CALL(hipMemset(slice, 0, sliceSizeMemWithPadding));
    auto *buffer = (double *) nvshmem_malloc(sliceSizeMemWithPadding);
    CUDA_SAFE_CALL(hipMemset(buffer, 0, sliceSizeMemWithPadding));

    auto commSync = (uint64_t *) nvshmem_malloc(2 * sizeof(uint64_t));
    CUDA_SAFE_CALL(hipMemset(commSync, 0, 2 * sizeof(uint64_t)));

    auto kernelSync = (uint64_t *) nvshmem_malloc(2 * sizeof(uint64_t));
    CUDA_SAFE_CALL(hipMemset(kernelSync, 0, 2 * sizeof(uint64_t)));

    // memory needed for verification
    double *unpaddedInputSlice;
    double *unpaddedOutputSlice;
    double *aggregatedInput;
    double *aggregatedResult;
    double *computeBuffer;

    if (verifyResults) {
        size_t sliceSizeMem = sliceSize.x * sliceSize.y * sliceSize.z * sizeof(double);
        size_t totalMem = SIZE * sizeof(double);

        unpaddedInputSlice = (double *) nvshmem_malloc(sliceSizeMem);
        unpaddedOutputSlice = (double *) nvshmem_malloc(sliceSizeMem);

        aggregatedInput = (double *) nvshmem_malloc(totalMem);
        aggregatedResult = (double *) nvshmem_malloc(totalMem);
        computeBuffer = (double *) nvshmem_malloc(totalMem);
        CUDA_SAFE_CALL(hipMemset(computeBuffer, 0, totalMem));
    }

    // create pseudo-random number generator
    hiprandState *random;
    CUDA_SAFE_CALL(hipMalloc(&random, sizeof(hiprandState)));
    setupRandom<<<1, 1, 0, stream>>>(random, worldRank);

    // init the data on every gpu
    initSlice<<<grid, threads, 0, stream>>>(slice, random, sliceSize);
    CUDA_SAFE_CALL(hipStreamSynchronize(stream));

    // send all slices to one GPU for verification
    if (verifyResults) {
        // remove padding
        removePadding<<<grid, threads, 0, stream>>>(slice, unpaddedInputSlice, sliceSize);
        hipStreamSynchronize(stream);

        // gather to GPU 0
        nvshmem_double_fcollect(NVSHMEM_TEAM_WORLD, aggregatedInput, unpaddedInputSlice, sliceSize.x * sliceSize.y * sliceSize.z);
    }

    // start measure execution time
    hipEvent_t start, end;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&end));

    nvshmem_barrier_all();

    for (int i = 0; i < ITERATIONS; i++) {

        if (i == 10) {
            CUDA_SAFE_CALL(hipEventRecord(start, stream));
        }

        laplaceInnerKernel<<<innerGrid, threads, 0, stream>>>(slice, buffer, sliceSize);

        if (worldRank != 0) {
            nvshmem_double_get(slice, &slice[paddedSliceSize.x * paddedSliceSize.y * (paddedSliceSize.z - 2)], paddedSliceSize.x * paddedSliceSize.y, worldRank - 1);
        }

        if (worldRank != worldSize - 1) {
            nvshmem_double_get(&slice[paddedSliceSize.x * paddedSliceSize.y * (paddedSliceSize.z - 1)], &slice[paddedSliceSize.x * paddedSliceSize.y], paddedSliceSize.x * paddedSliceSize.y, worldRank + 1);
        }

        sync_neighbours<<<1, 1, 0, stream>>>(worldRank, worldSize, commSync, i + 1);

        laplaceOuterKernel<<<outerGrid, threads, 0, stream>>>(slice, buffer, sliceSize);

        sync_neighbours<<<1, 1, 0, stream>>>(worldRank, worldSize, kernelSync, i + 1);

        // swap buffers
        std::swap(slice, buffer);
    }

    // end measure execution time
    CUDA_SAFE_CALL(hipEventRecord(end, stream));
    CUDA_SAFE_CALL(hipEventSynchronize(end));

    nvshmem_barrier_all();

    float milliseconds;
    CUDA_SAFE_CALL(hipEventElapsedTime(&milliseconds, start, end));

    auto *maxExecutionTime = (float *) nvshmem_malloc(sizeof(float));
    CUDA_SAFE_CALL(hipMemcpy(maxExecutionTime, &milliseconds, sizeof(float), hipMemcpyHostToDevice));
    nvshmemx_float_max_reduce_on_stream(NVSHMEM_TEAM_WORLD, maxExecutionTime, maxExecutionTime, 1, stream);
    
    if (worldRank == 0) {
        printTimings<<<1, 1, 0, stream>>>(maxExecutionTime);
    }
    hipStreamSynchronize(stream);

    // verify results on one GPU
    if (verifyResults) {
        // remove padding
        removePadding<<<grid, threads, 0, stream>>>(slice, unpaddedOutputSlice, sliceSize);
        CUDA_SAFE_CALL(hipStreamSynchronize(stream));

        // gather to GPU 0
        nvshmem_double_fcollect(NVSHMEM_TEAM_WORLD, aggregatedResult, unpaddedOutputSlice, sliceSize.x * sliceSize.y * sliceSize.z);

        // compute on single GPU and compare
        if (worldRank == 0) {
            // need new grid dimensions for total grid
            dim3 total, totalGrid;
            total.x = WIDTH;
            total.y = HEIGHT;
            total.z = DEPTH;

            totalGrid.x = std::ceil((float) total.x / threads.x);
            totalGrid.y = std::ceil((float) total.y / threads.y);
            totalGrid.z = std::ceil((float) total.z / threads.z);

            for (int i = 0; i < ITERATIONS; i++) {
                // swap buffers
                std::swap(aggregatedInput, computeBuffer);

                // laplace step
                laplace_singleGPU<<<totalGrid, threads, 0, stream>>>(aggregatedInput, computeBuffer, total);
                CUDA_SAFE_CALL(hipStreamSynchronize(stream));
            }

            // compare
            verify<<<totalGrid, threads, 0, stream>>>(aggregatedInput, aggregatedResult, total);
            CUDA_SAFE_CALL(hipStreamSynchronize(stream));

            // verification done
            printf("result correctness verified.\n");
        }
    }

    // cleanup
    nvshmem_free(slice);
    nvshmem_free(buffer);

    nvshmem_free(commSync);
    nvshmem_free(kernelSync);

    if (verifyResults) {
        nvshmem_free(unpaddedInputSlice);
        nvshmem_free(unpaddedOutputSlice);
        nvshmem_free(aggregatedInput);
        nvshmem_free(aggregatedResult);
        nvshmem_free(computeBuffer);
    }

    nvshmem_finalize();
    return 0;
}