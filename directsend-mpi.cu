#include "hip/hip_runtime.h"
#include <iostream>
#include <mpi.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_SAFE_CALL(call) \
do { \
    hipError_t err = call; \
    if (hipSuccess != err) { \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, __LINE__, hipGetErrorString(err) ); \
        exit(EXIT_FAILURE); \
    } \
} while (0)                  \

const int IMAGE_WIDTH = 960;
const int IMAGE_HEIGHT = 960;
const int IMAGE_SIZE = IMAGE_WIDTH * IMAGE_HEIGHT;

const int RAY_ARRAY_LENGTH = 2048;
const int ITERATIONS = 110;

__device__
int indexAt(int x, int y) {
    return x * IMAGE_HEIGHT + y;
}

__global__
void setupRandom(hiprandState *random, int seed) {
    hiprand_init(seed, 0, 0, random);
}

__global__
void initRayArray(float *rayArray, hiprandState *random, int sliceSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < sliceSize) {
        for (int i = 0; i < RAY_ARRAY_LENGTH; i++) {
            rayArray[index * RAY_ARRAY_LENGTH + i] = hiprand_uniform(random);
        }
    }
}

__global__
void computeImage(float *image, const float *rayArray, int worldRank, int sliceWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x / sliceWidth == worldRank) {
        float accumulated = 0;
        for (int i = 0; i < RAY_ARRAY_LENGTH; i++) {
            accumulated += rayArray[indexAt(x,y) * RAY_ARRAY_LENGTH + i];
        }

        image[indexAt(x,y)] = accumulated;
    } else {
        image[indexAt(x,y)] = 0;
    }
}

__global__
void composite(float *compositedSlice, const float *slices, int worldSize, int sliceWidth) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    for (int i = 0; i < worldSize; i++) {
        sum += slices[indexAt(i * sliceWidth + x, y)];
    }

    compositedSlice[indexAt(x,y)] = sum;
}

int main(int argc, char *argv[]) {

    MPI_Init(nullptr, nullptr);

    int worldSize;
    MPI_Comm_size(MPI_COMM_WORLD, &worldSize);

    int worldRank;
    MPI_Comm_rank(MPI_COMM_WORLD, &worldRank);

    MPI_Comm nodeCommunicator;
    MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL, &nodeCommunicator);
    int nodeRank;
    MPI_Comm_rank(nodeCommunicator, &nodeRank);

    CUDA_SAFE_CALL(hipSetDevice(nodeRank));

    hipStream_t stream;
    CUDA_SAFE_CALL(hipStreamCreate(&stream));

    int sliceSize = IMAGE_SIZE / worldSize;

    // only needed on rank 0
    float *result;

    // device memory
    float *image1;
    float *image2;
    float *slices;
    float *compositedSlice;

    float *rayArray;

    if (worldRank == 0) {
        CUDA_SAFE_CALL(hipMalloc(&result, IMAGE_SIZE * sizeof(float)));
    }

    CUDA_SAFE_CALL(hipMalloc(&image1, IMAGE_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMemset(image1, 0, IMAGE_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&image2, IMAGE_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMemset(image2, 0, IMAGE_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&slices, IMAGE_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&compositedSlice, sliceSize * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc(&rayArray, IMAGE_SIZE * RAY_ARRAY_LENGTH * sizeof(float)));

    // create pseudo-random number generator
    hiprandState *random;
    CUDA_SAFE_CALL(hipMalloc(&random, sizeof(hiprandState)));
    setupRandom<<<1, 1, 0, stream>>>(random, worldRank);

    initRayArray<<<std::ceil((float) sliceSize / 1024), 1024, 0, stream>>>(rayArray, random, sliceSize);
    CUDA_SAFE_CALL(hipStreamSynchronize(stream));

    hipEvent_t start, end;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&end));

    MPI_Barrier(MPI_COMM_WORLD);

    int sliceWidth = IMAGE_WIDTH / worldSize;

    dim3 threads(1, IMAGE_HEIGHT, 1);
    dim3 imageGrid(IMAGE_WIDTH, 1, 1);
    dim3 sliceGrid(sliceWidth, 1, 1);

    // compute image 1
    computeImage<<<imageGrid, threads, 0, stream>>>(image1, rayArray, worldRank, sliceWidth);
    hipStreamSynchronize(stream);

    for (int i = 0; i < ITERATIONS; i++) {

        if (i == 10) {
            CUDA_SAFE_CALL(hipEventRecord(start, stream));
        }

        // compute image 2
        computeImage<<<imageGrid, threads, 0, stream>>>(image2, rayArray, worldRank, sliceWidth);

        // while image 2 is computing, distribute image 1 parts to the respective ranks
        MPI_Alltoall(image1, sliceSize, MPI_FLOAT, slices, sliceSize, MPI_FLOAT, MPI_COMM_WORLD);

        // composite the slices of image 1
        composite<<<sliceGrid, threads, 0, stream>>>(compositedSlice, slices, worldSize, sliceWidth);
        CUDA_SAFE_CALL(hipStreamSynchronize(stream));

        // gather the composited slices of image 1 to rank 0
        MPI_Gather(compositedSlice, sliceSize, MPI_FLOAT, result, sliceSize, MPI_FLOAT, 0, MPI_COMM_WORLD);

        // swap buffers
        std::swap(image1, image2);
    }

    // distribute the last image
    MPI_Alltoall(image1, sliceSize, MPI_FLOAT, slices, sliceSize, MPI_FLOAT, MPI_COMM_WORLD);

    composite<<<sliceGrid, threads, 0, stream>>>(compositedSlice, slices, worldSize, sliceWidth);
    CUDA_SAFE_CALL(hipStreamSynchronize(stream));

    MPI_Gather(compositedSlice, sliceSize, MPI_FLOAT, result, sliceSize, MPI_FLOAT, 0, MPI_COMM_WORLD);

    // done.

    CUDA_SAFE_CALL(hipEventRecord(end, stream));
    CUDA_SAFE_CALL(hipEventSynchronize(end));

    MPI_Barrier(MPI_COMM_WORLD);

    float executionTime;
    CUDA_SAFE_CALL(hipEventElapsedTime(&executionTime, start, end));

    float maxExecutionTime = 0;
    MPI_Reduce(&executionTime, &maxExecutionTime, 1, MPI_FLOAT, MPI_MAX, 0, MPI_COMM_WORLD);

    if (worldRank == 0) {
        printf("execution done.\n max execution time: %f ms\n", maxExecutionTime);
    }

    if (worldRank == 0) {
        CUDA_SAFE_CALL(hipFree(result));
    }

    CUDA_SAFE_CALL(hipFree(image1));
    CUDA_SAFE_CALL(hipFree(image2));
    CUDA_SAFE_CALL(hipFree(slices));
    CUDA_SAFE_CALL(hipFree(compositedSlice));
    CUDA_SAFE_CALL(hipFree(rayArray));

    MPI_Finalize();
    return 0;
}